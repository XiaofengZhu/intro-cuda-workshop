
#include <hip/hip_runtime.h>
#include <cmath>
#include <cstdio>
#include <ctime>
#include <iostream>

// Define Kernel to run on device (GPU)
__global__
void add(float *d_a, float *d_b, float *d_c, long num)
{
    for (long ii = 0; ii < num; ++ii) {
        d_c[ii] = d_a[ii] + d_b[ii];
    }
}

int main(void)
{
 
    std::clock_t start_time;
    double duration01;
    double duration02;
    double duration03;
    
    // Define array size and memory
    const long ARR_SIZE =  500000000;
    const size_t ARR_BYTES =  ARR_SIZE*sizeof(float);

    // Clock start
    start_time = std::clock();
    
    // Declare and alloc array on host
    float h_a[ARR_SIZE];
    float h_b[ARR_SIZE];
    float h_c[ARR_SIZE];

    // Initialize array elements
    for (long i=0;  i<ARR_SIZE; i++){
        h_a[i] = float(i);
        h_b[i] = float(i)*2.0;
    }

    // Declare and alloc array on device
    float *d_a;
    float *d_b;
    float *d_c;
    hipMalloc(&d_a, ARR_BYTES);
    hipMalloc(&d_b, ARR_BYTES);
    hipMalloc(&d_c, ARR_BYTES);

    // Transfer to device
    hipMemcpy(d_a, h_a, ARR_BYTES, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, ARR_BYTES, hipMemcpyHostToDevice);

    // Clock stop 01
    duration01 = ( std::clock() - start_time ) / (double) CLOCKS_PER_SEC;
    std::cout<<"Computing time until Kernel call: "<< duration01 << "s" << std::endl;

    // Call kernel function
    add<<<1, 1>>>(d_a, d_b, d_c, ARR_SIZE);

    // Block until the device has completed all tasks
    hipDeviceSynchronize();

    // Clock stop 02
    duration02 = ( std::clock() - start_time ) / (double) CLOCKS_PER_SEC;
    std::cout<<"Computing time after Kernel call: "<< duration02 << "s" << std::endl;

    // Transfer results to host
    hipMemcpy(h_c, d_c, ARR_BYTES, hipMemcpyDeviceToHost);

    // Clock stop 03
    duration03 = ( std::clock() - start_time ) / (double) CLOCKS_PER_SEC;
    std::cout<<"Computing time after memory copy: "<< duration03 << "s" << std::endl;

    // Output results
    for(long ii=0; ii<10; ii++){
        std::cout<< h_c[ii]<< ", ";
    }    
    std::cout<< std::endl;

    return 0;
}
