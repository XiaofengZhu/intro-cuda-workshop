
#include <hip/hip_runtime.h>
#include <cmath>
#include <cstdio>
#include <ctime>
#include <iostream>

__global__
void add(float *d_a, float *d_b, float *d_c, long num)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < num) {
        d_c[idx] = d_a[idx] + d_b[idx];
    }
}

int main(void)
{
    std::clock_t start_time;
    double duration01;
    double duration02;
    double duration03;

    const long ARR_SIZE =  500000000;
    const size_t ARR_BYTES =  ARR_SIZE*sizeof(float);

    // Clock start
    start_time = std::clock();
    
    // Declare and alloc array on host
    float h_a[ARR_SIZE];
    float h_b[ARR_SIZE];
    float h_c[ARR_SIZE];

    // initialize input array 
    for (long i=0;  i<ARR_SIZE; i++){
        h_a[i] = float(i);
        h_b[i] = float(i)*2.0;
    }

    // Declare and alloc array on device
    float *d_a;
    float *d_b;
    float *d_c;
    hipMalloc(&d_a, ARR_BYTES);
    hipMalloc(&d_b, ARR_BYTES);
    hipMalloc(&d_c, ARR_BYTES);

    // Transfer to device
    hipMemcpy(d_a, h_a, ARR_BYTES, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, ARR_BYTES, hipMemcpyHostToDevice);

    // Clock stop 01
    duration01 = ( std::clock() - start_time ) / (double) CLOCKS_PER_SEC;
    std::cout<<"Computing time until Kernel call: "<< duration01 << "s" << std::endl;

    // Call kernel function
    const int threadPerBlock = 1024;
    const int numBlock = ARR_SIZE/threadPerBlock+1;
    add<<<numBlock, threadPerBlock>>>(d_a, d_b, d_c, ARR_SIZE);

    // Block until the device has completed all tasks
    hipDeviceSynchronize();

    // Clock stop 02
    duration02 = ( std::clock() - start_time ) / (double) CLOCKS_PER_SEC;
    std::cout<<"Computing time after Kernel call: "<< duration02 << "s" << std::endl;

    // Transfer results to host
    hipMemcpy(h_c, d_c, ARR_BYTES, hipMemcpyDeviceToHost);

    // Clock stop 03
    duration03 = ( std::clock() - start_time ) / (double) CLOCKS_PER_SEC;
    std::cout<<"Computing time after memory copy: "<< duration03 << "s" << std::endl;

    // Output results
    for(long ii=0; ii<10; ii++){
        std::cout<< h_c[ii]<< ", ";
    }    
    std::cout<< std::endl;

    return 0;
}
